#include "hip/hip_runtime.h"
/*
 * Software License Agreement (BSD License)
 *
 *  Point Cloud Library (PCL) - www.pointclouds.org
 *  Copyright (c) 2011, Willow Garage, Inc.
 *
 *  All rights reserved.
 *
 *  Redistribution and use in source and binary forms, with or without
 *  modification, are permitted provided that the following conditions
 *  are met:
 *
 *   * Redistributions of source code must retain the above copyright
 *     notice, this list of conditions and the following disclaimer.
 *   * Redistributions in binary form must reproduce the above
 *     copyright notice, this list of conditions and the following
 *     disclaimer in the documentation and/or other materials provided
 *     with the distribution.
 *   * Neither the name of Willow Garage, Inc. nor the names of its
 *     contributors may be used to endorse or promote products derived
 *     from this software without specific prior written permission.
 *
 *  THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS
 *  "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
 *  LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS
 *  FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE
 *  COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT,
 *  INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
 *  BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 *  LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 *  CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT
 *  LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN
 *  ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 *  POSSIBILITY OF SUCH DAMAGE.
 *
 */

#include "device.hpp"
//#include <pcl/gpu/utils/device/block.hpp>
#include <pcl/gpu/containers/device_array.h>

#include <thrust/extrema.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/execution_policy.h>

#include <thrust/device_vector.h>
#include <thrust/transform.h>
#include <thrust/iterator/zip_iterator.h>
#include <stdio.h>

namespace pcl
{
namespace device
{
__device__ unsigned int count = 0;

struct CorespSearch
{
    enum { CTA_SIZE_X = 32, CTA_SIZE_Y = 8, CTA_SIZE = CTA_SIZE_X * CTA_SIZE_Y };

    struct plus
    {
        __forceinline__ __device__ int
        operator () (const int &lhs, const volatile int& rhs) const {
            return lhs + rhs;
        }
    };


    PtrStep<float> vmap_g_curr;
    PtrStep<float> nmap_g_curr;

    Mat33 Rprev_inv;
    float3 tprev;

    Intr intr;

    PtrStep<float> vmap_g_prev;
    PtrStep<float> nmap_g_prev;

    float distThres;
    float angleThres;

    mutable int *dist_PDF;
    mutable int *dist_CPDF;
    mutable int dist_median_bin;
    mutable float dist_median_value;
    mutable int *Output;

    mutable float RangeDistHist; //m
    float range_around_median; //m

    mutable PtrStepSz<short2> coresp;
    mutable PtrStepSz<float> dist_matr;

    mutable int* gbuf;
    int nb_bin;




    mutable PtrStepSz<bool> error_spotted;



    __device__ __forceinline__ int
    estimateDistances() const
    {
        int x = threadIdx.x + blockIdx.x * blockDim.x;
        int y = threadIdx.y + blockIdx.y * blockDim.y;

        if (x >= coresp.cols || y >= coresp.rows)
            return 0;

        error_spotted.ptr(y)[x] = true;
        dist_matr.ptr (y)[x] = -1;
        coresp.ptr (y)[x] = make_short2 (-1, -1);

        float3 ncurr_g;
        ncurr_g.x = nmap_g_curr.ptr (y)[x];

        if (isnan (ncurr_g.x))
        {
            error_spotted.ptr(y)[x] = true;
            return 0;
        }
        float3 vcurr_g;
        vcurr_g.x = vmap_g_curr.ptr (y              )[x];
        vcurr_g.y = vmap_g_curr.ptr (y + coresp.rows)[x];
        vcurr_g.z = vmap_g_curr.ptr (y + 2 * coresp.rows)[x];

        float3 vcurr_cp = Rprev_inv * (vcurr_g - tprev);         // prev camera coo space

        int2 ukr;         //projection
        ukr.x = __float2int_rn (vcurr_cp.x * intr.fx / vcurr_cp.z + intr.cx);      //4
        ukr.y = __float2int_rn (vcurr_cp.y * intr.fy / vcurr_cp.z + intr.cy);                      //4

        if (ukr.x < 0 || ukr.y < 0 || ukr.x >= coresp.cols || ukr.y >= coresp.rows)
        {
            error_spotted.ptr(y)[x] = true;
            return 0;
        }

        float3 nprev_g;
        nprev_g.x = nmap_g_prev.ptr (ukr.y)[ukr.x];

        if (isnan (nprev_g.x))
        {
            error_spotted.ptr(y)[x] = true;
            return 0;
        }

        float3 vprev_g;
        vprev_g.x = vmap_g_prev.ptr (ukr.y              )[ukr.x];
        vprev_g.y = vmap_g_prev.ptr (ukr.y + coresp.rows)[ukr.x];
        vprev_g.z = vmap_g_prev.ptr (ukr.y + 2 * coresp.rows)[ukr.x];


        float dist = norm (vcurr_g - vprev_g);
        dist_matr.ptr (y)[x] = dist;




        ncurr_g.y = nmap_g_curr.ptr (y + coresp.rows)[x];
        ncurr_g.z = nmap_g_curr.ptr (y + 2 * coresp.rows)[x];


        nprev_g.y = nmap_g_prev.ptr (ukr.y + coresp.rows)[ukr.x];
        nprev_g.z = nmap_g_prev.ptr (ukr.y + 2 * coresp.rows)[ukr.x];

        float sine = norm (cross (ncurr_g, nprev_g));

        /*if (sine >= 1 || asinf(sine) >= angleThres)
            return 0;*/

        if (/*sine >= 1 || */ sine >= angleThres)
        {
            error_spotted.ptr(y)[x] = true;
            return 0;
        }


        error_spotted.ptr(y)[x] = false;
        return 1;
    }


    __device__ __forceinline__ int
    fill_PDF_histogram() const
    {
        int x = threadIdx.x + blockIdx.x * blockDim.x;
        int y = threadIdx.y + blockIdx.y * blockDim.y;
        if (x >= coresp.cols || y >= coresp.rows)
            return 0;

        if ( error_spotted.ptr(y)[x])
            return 0;

        float resolution = RangeDistHist / nb_bin;

        int i_hist = dist_matr.ptr (y)[x] / resolution;

        if (i_hist < 0 || i_hist >= nb_bin)
        {
            error_spotted.ptr(y)[x] = true;
            return 0;
        }
        atomicAdd(dist_PDF + i_hist, 1);

        return 1;
    }



    // 1 thread only
    __device__ __forceinline__ int
    fill_CPDF_histogram() const
    {

        // int tid = Block::flattenedThreadId();
        // printf ("ThreadId for prefixSum is (should be 0, and appear once) : %d\n", tid);

        dist_CPDF[0] = dist_PDF[0];
        for (int i = 1; i < nb_bin; i++)
        {
            dist_CPDF[i] = dist_CPDF[i-1] + dist_PDF[i];
            //  printf ("   -> %d th element of CFPD : %d\n", i, dist_CPDF[i]);
        }
        return 1;
    }



    // 1000 thread
    __device__ __forceinline__ int
    estimateMedian() const
    {

        int tid = Block::flattenedThreadId();

        if (tid == 0)
        {
            int NbCorrepSoFar = dist_CPDF[tid];
            int NbCorrep_ = dist_CPDF[nb_bin-1];
            if (NbCorrepSoFar >= NbCorrep_/2)
            {
                dist_median_bin = tid;
              //  dist_median_value = tid * RangeDistHist / nb_bin;

                int bin_end = dist_median_bin + __float2int_rn(range_around_median * nb_bin / RangeDistHist);

                Output[0] = dist_CPDF [ bin_end ];
                Output[1] = dist_median_bin;


            }
        }
        else if (tid > 0)
        {
            int NbCorrepPrevious = dist_CPDF[tid-1];
            int NbCorrepSoFar = dist_CPDF[tid];
            int NbCorrep_ = dist_CPDF[nb_bin-1];
            if (NbCorrepPrevious < NbCorrep_/2 && NbCorrepSoFar >= NbCorrep_/2)
            {
                dist_median_bin = tid;
                dist_median_value = tid * RangeDistHist / nb_bin;
                //printf ("Median estimation : %f of bin %d\n", dist_median_value, dist_median_bin);
                int bin_init = dist_median_bin - __float2int_rn(range_around_median * nb_bin / RangeDistHist);
                int bin_end = dist_median_bin + __float2int_rn(range_around_median * nb_bin / RangeDistHist);

                if (bin_init < 0) bin_init = 0;
                if (bin_end > nb_bin-1) bin_end = nb_bin-1;
                Output[0] = dist_CPDF [ bin_end ] - dist_CPDF [ bin_init ];
                Output[1] = dist_median_bin;

            }
        }
        return 1;

    }


    // 300k thread
    __device__ __forceinline__ int
    findFilteredCorrespondences () const
    {
        int x = threadIdx.x + blockIdx.x * blockDim.x;
        int y = threadIdx.y + blockIdx.y * blockDim.y;


        /// HERE I CAN ADD A FILTERING BASED ON % OF POINT I HAVE LEFT!!
        ///
        /*  if (Output[0] < 0.1 * dist_CPDF[nb_bin-1])
        {
            error_spotted.ptr(y)[x] = true;
            return 0;
        }*/



        if ( error_spotted.ptr(y)[x])
            return 0;


        if (dist_matr.ptr (y)[x] < 0)
        {

            printf("    ->    How come dist is negatif and the error has not been spotted yet??   -->> PLEASE DEBUG ME IF YOU SEE THIS!!!!");
            error_spotted.ptr(y)[x] = true;
            return 0;
        }



        if (dist_matr.ptr (y)[x] > dist_median_value + range_around_median)
        {
            error_spotted.ptr(y)[x] = true;
            return 0;
        }


        if (dist_matr.ptr (y)[x] < dist_median_value - range_around_median)
        {
            error_spotted.ptr(y)[x] = true;
            return 0;
        }





        float3 vcurr_g;
        vcurr_g.x = vmap_g_curr.ptr (y              )[x];
        vcurr_g.y = vmap_g_curr.ptr (y + coresp.rows)[x];
        vcurr_g.z = vmap_g_curr.ptr (y + 2 * coresp.rows)[x];

        float3 vcurr_cp = Rprev_inv * (vcurr_g - tprev);         // prev camera coo space
        int2 ukr;         //projection
        ukr.x = __float2int_rn (vcurr_cp.x * intr.fx / vcurr_cp.z + intr.cx);      //4
        ukr.y = __float2int_rn (vcurr_cp.y * intr.fy / vcurr_cp.z + intr.cy);                      //4



        coresp.ptr (y)[x] = make_short2 (ukr.x, ukr.y);


        return 1;
    }


    __device__ __forceinline__ int
    search () const
    {

        int x = threadIdx.x + blockIdx.x * blockDim.x;
        int y = threadIdx.y + blockIdx.y * blockDim.y;

        if (x >= coresp.cols || y >= coresp.rows)
            return 0;

        coresp.ptr (y)[x] = make_short2 (-1, -1);

        float3 ncurr_g;
        ncurr_g.x = nmap_g_curr.ptr (y)[x];

        if (isnan (ncurr_g.x))
            return 0;

        float3 vcurr_g;
        vcurr_g.x = vmap_g_curr.ptr (y              )[x];
        vcurr_g.y = vmap_g_curr.ptr (y + coresp.rows)[x];
        vcurr_g.z = vmap_g_curr.ptr (y + 2 * coresp.rows)[x];

        float3 vcurr_cp = Rprev_inv * (vcurr_g - tprev);         // prev camera coo space

        int2 ukr;         //projection
        ukr.x = __float2int_rn (vcurr_cp.x * intr.fx / vcurr_cp.z + intr.cx);      //4
        ukr.y = __float2int_rn (vcurr_cp.y * intr.fy / vcurr_cp.z + intr.cy);                      //4

        if (ukr.x < 0 || ukr.y < 0 || ukr.x >= coresp.cols || ukr.y >= coresp.rows)
            return 0;

        float3 nprev_g;
        nprev_g.x = nmap_g_prev.ptr (ukr.y)[ukr.x];

        if (isnan (nprev_g.x))
            return 0;

        float3 vprev_g;
        vprev_g.x = vmap_g_prev.ptr (ukr.y              )[ukr.x];
        vprev_g.y = vmap_g_prev.ptr (ukr.y + coresp.rows)[ukr.x];
        vprev_g.z = vmap_g_prev.ptr (ukr.y + 2 * coresp.rows)[ukr.x];

        float dist = norm (vcurr_g - vprev_g);
        if (dist > distThres)
            return 0;

        ncurr_g.y = nmap_g_curr.ptr (y + coresp.rows)[x];
        ncurr_g.z = nmap_g_curr.ptr (y + 2 * coresp.rows)[x];

        nprev_g.y = nmap_g_prev.ptr (ukr.y + coresp.rows)[ukr.x];
        nprev_g.z = nmap_g_prev.ptr (ukr.y + 2 * coresp.rows)[ukr.x];

        float sine = norm (cross (ncurr_g, nprev_g));

        /*if (sine >= 1 || asinf(sine) >= angleThres)
                    return 0;*/

        if (/*sine >= 1 || */ sine >= angleThres)
            return 0;

        coresp.ptr (y)[x] = make_short2 (ukr.x, ukr.y);
        return 1;
    }
/*
    __device__ __forceinline__ void
    operator () () const
    {
        int mask = search ();

    }*/
};

__global__ void
corespKernel (const CorespSearch cs) {
    cs.search();
}

__global__ void
corespFinding (const CorespSearch cs) {
    cs.estimateDistances();
}

__global__ void
corespFiltering (const CorespSearch cs) {
    cs.findFilteredCorrespondences();
}

__global__ void
histFilling (const CorespSearch cs) {
    cs.fill_PDF_histogram();
}

__global__ void
CPDF_Filling (const CorespSearch cs) {
    cs.fill_CPDF_histogram();
}


__global__ void
estimateMedian (const CorespSearch cs) {
    cs.estimateMedian();
}



}
}

//////////////////////////////////////////////////////////////////////////////////////////////////////////////////
int
pcl::device::findCoresp (const MapArr& vmap_g_curr, const MapArr& nmap_g_curr, 
                         const Mat33& Rprev_inv, const float3& tprev, const Intr& intr,
                         const MapArr& vmap_g_prev, const MapArr& nmap_g_prev,
                         float distThres, float angleThres, PtrStepSz<short2> coresp/*, float* median*/)
{
    // Matrix of distances
    pcl::gpu::DeviceArray2D<float> dist_matr;
    dist_matr.create(coresp.rows, coresp.cols);



    //PDF and CPDF of distances
    int nbBin = 1000;
    float RangeDistHist = 0.5f;
    float range_around_median = 0.1f;



    pcl::gpu::DeviceArray<int> dist_PDF;
    pcl::gpu::DeviceArray<int> dist_CPDF;
    int dist_PDF_host[nbBin];
    int dist_CPDF_host[nbBin];
    for (int i = 0; i < nbBin; i++)
    {
        dist_PDF_host[i] = 0;
        dist_CPDF_host[i] = 0;
    }

    dist_PDF.upload(dist_PDF_host, nbBin);
    dist_CPDF.upload(dist_CPDF_host, nbBin);

    // Matrix of error bool
    pcl::gpu::DeviceArray2D<bool> error_spotted;
    error_spotted.create(coresp.rows, coresp.cols);



    CorespSearch cs;

    cs.vmap_g_curr = vmap_g_curr;
    cs.nmap_g_curr = nmap_g_curr;

    cs.Rprev_inv = Rprev_inv;
    cs.tprev = tprev;

    cs.intr = intr;

    cs.vmap_g_prev = vmap_g_prev;
    cs.nmap_g_prev = nmap_g_prev;

    cs.distThres = distThres;
    cs.angleThres = angleThres;

    cs.coresp = coresp;

    cs.nb_bin = nbBin;
    cs.dist_PDF = dist_PDF;
    cs.dist_CPDF = dist_CPDF;
    cs.dist_matr = dist_matr;
    cs.error_spotted = error_spotted;

    cs.range_around_median = range_around_median;
    cs.RangeDistHist = RangeDistHist;



    pcl::gpu::DeviceArray<int> output_device;
    output_device.create(2);
    // output_device[0] = coresp.rows * coresp.cols;
    cs.Output = output_device;

    dim3 block (CorespSearch::CTA_SIZE_X, CorespSearch::CTA_SIZE_Y);
    dim3 grid (divUp (coresp.cols, block.x), divUp (coresp.rows, block.y));



    // Resutls
    int output[2];
    output[0] = 1;
    output[1] = 1;



    /*
    corespKernel<<<grid, block>>>(cs);

    cudaSafeCall ( hipGetLastError () );
    cudaSafeCall (hipDeviceSynchronize ());
*/





    corespFinding<<<grid, block>>>(cs);

    cudaSafeCall ( hipGetLastError () );
    cudaSafeCall ( hipDeviceSynchronize ());


    //300k thread -> each pixel/dist
    histFilling<<<grid, block>>>(cs);

    cudaSafeCall ( hipGetLastError () );
    cudaSafeCall ( hipDeviceSynchronize ());


    // from PCF, get CPDF
    CPDF_Filling<<<1, 1>>>(cs);

    cudaSafeCall ( hipGetLastError () );
    cudaSafeCall ( hipDeviceSynchronize ());

    // Estiamte Median on CPDF -> 1000 threads
    estimateMedian<<<1, nbBin>>>(cs);

    cudaSafeCall ( hipGetLastError () );
    cudaSafeCall ( hipDeviceSynchronize ());


    // Filter correspondences -> 300k thread
    corespFiltering<<<grid, block>>>(cs);

    cudaSafeCall ( hipGetLastError () );
    cudaSafeCall ( hipDeviceSynchronize ());


    output_device.download(output);

  //  median[0] = output[1]*RangeDistHist/nbBin;
    std::cout << "Median estimation : " << output[1]*RangeDistHist/nbBin << " of bin " << output[1] << std::endl;
    std::cout << "Percentage of matches = " << (output[0]*1.f)/(coresp.cols*coresp.rows)
              << "(" << output[0] << "/" << coresp.cols*coresp.rows << ")" << std::endl;


    return output[1];

}
