#include "hip/hip_runtime.h"
/*
 * Software License Agreement (BSD License)
 *
 *  Point Cloud Library (PCL) - www.pointclouds.org
 *  Copyright (c) 2011, Willow Garage, Inc.
 *
 *  All rights reserved.
 *
 *  Redistribution and use in source and binary forms, with or without
 *  modification, are permitted provided that the following conditions
 *  are met:
 *
 *   * Redistributions of source code must retain the above copyright
 *     notice, this list of conditions and the following disclaimer.
 *   * Redistributions in binary form must reproduce the above
 *     copyright notice, this list of conditions and the following
 *     disclaimer in the documentation and/or other materials provided
 *     with the distribution.
 *   * Neither the name of Willow Garage, Inc. nor the names of its
 *     contributors may be used to endorse or promote products derived
 *     from this software without specific prior written permission.
 *
 *  THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS
 *  "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
 *  LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS
 *  FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE
 *  COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT,
 *  INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
 *  BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 *  LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 *  CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT
 *  LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN
 *  ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 *  POSSIBILITY OF SUCH DAMAGE.
 *
 */

//#include <pcl/gpu/utils/device/block.hpp>
//#include <pcl/gpu/utils/device/funcattrib.hpp>
#include "device.hpp"

#include <pcl/gpu/containers/device_array.h>

#include <thrust/extrema.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/execution_policy.h>

#include <thrust/device_vector.h>
#include <thrust/transform.h>
#include <thrust/iterator/zip_iterator.h>
#include <stdio.h>

namespace pcl
{
namespace device
{
typedef float float_type;

template<int CTA_SIZE_, typename T>
static __device__ __forceinline__ void reduce(volatile T* buffer)
{
    int tid = Block::flattenedThreadId();
    T val =  buffer[tid];

    if (CTA_SIZE_ >= 1048576) { if (tid < 524288) buffer[tid] = val = val + buffer[tid + 524288]; __syncthreads(); }
    if (CTA_SIZE_ >= 524288) { if (tid < 262144) buffer[tid] = val = val + buffer[tid + 262144]; __syncthreads(); }
    if (CTA_SIZE_ >= 262144) { if (tid < 131072) buffer[tid] = val = val + buffer[tid + 131072]; __syncthreads(); }
    if (CTA_SIZE_ >= 131072) { if (tid < 65536) buffer[tid] = val = val + buffer[tid + 65536]; __syncthreads(); }
    if (CTA_SIZE_ >= 65536) { if (tid < 32768) buffer[tid] = val = val + buffer[tid + 32768]; __syncthreads(); }
    if (CTA_SIZE_ >= 32768) { if (tid < 16384) buffer[tid] = val = val + buffer[tid + 16384]; __syncthreads(); }
    if (CTA_SIZE_ >= 16384) { if (tid < 8192) buffer[tid] = val = val + buffer[tid + 8192]; __syncthreads(); }
    if (CTA_SIZE_ >= 8192) { if (tid < 4096) buffer[tid] = val = val + buffer[tid + 4096]; __syncthreads(); }
    if (CTA_SIZE_ >= 4096) { if (tid < 2048) buffer[tid] = val = val + buffer[tid + 2048]; __syncthreads(); }
    if (CTA_SIZE_ >= 2048) { if (tid < 1024) buffer[tid] = val = val + buffer[tid + 1024]; __syncthreads(); }
    if (CTA_SIZE_ >= 1024) { if (tid < 512) buffer[tid] = val = val + buffer[tid + 512]; __syncthreads(); }
    if (CTA_SIZE_ >=  512) { if (tid < 256) buffer[tid] = val = val + buffer[tid + 256]; __syncthreads(); }
    if (CTA_SIZE_ >=  256) { if (tid < 128) buffer[tid] = val = val + buffer[tid + 128]; __syncthreads(); }
    if (CTA_SIZE_ >=  128) { if (tid <  64) buffer[tid] = val = val + buffer[tid +  64]; __syncthreads(); }

    if (tid < 32)
    {
        if (CTA_SIZE_ >=   64) { buffer[tid] = val = val + buffer[tid +  32]; }
        if (CTA_SIZE_ >=   32) { buffer[tid] = val = val + buffer[tid +  16]; }
        if (CTA_SIZE_ >=   16) { buffer[tid] = val = val + buffer[tid +   8]; }
        if (CTA_SIZE_ >=    8) { buffer[tid] = val = val + buffer[tid +   4]; }
        if (CTA_SIZE_ >=    4) { buffer[tid] = val = val + buffer[tid +   2]; }
        if (CTA_SIZE_ >=    2) { buffer[tid] = val = val + buffer[tid +   1]; }
    }
}

struct Combined
{
    enum
    {
        CTA_SIZE_X = 32,
        CTA_SIZE_Y = 8,
        CTA_SIZE = CTA_SIZE_X * CTA_SIZE_Y
    };


    Mat33 Rcurr;
    float3 tcurr;

    PtrStep<float> vmap_curr;
    PtrStep<float> nmap_curr;

    Mat33 Rprev_inv;
    float3 tprev;

    Intr intr;

    PtrStep<float> vmap_g_prev;
    PtrStep<float> nmap_g_prev;

    float distThres;
    float angleThres;

 /*   mutable int *dist_PDF;
    mutable int *dist_CPDF;
    mutable int dist_median_bin;
    mutable float dist_median_value;
    mutable int *Output;

    mutable float RangeDistHist; //m
    int nb_bin;*/

    int cols;
    int rows;
    mutable int* cnt_buffer;
    mutable PtrStep<float_type> dist_array;

    mutable PtrStep<float_type> gbuf;

    __device__ __forceinline__ bool
    search (int x, int y, float3& n, float3& d, float3& s) const
    {
        float3 ncurr;
        ncurr.x = nmap_curr.ptr (y)[x];

        if (isnan (ncurr.x))
            return (false);

        float3 vcurr;
        vcurr.x = vmap_curr.ptr (y       )[x];
        vcurr.y = vmap_curr.ptr (y + rows)[x];
        vcurr.z = vmap_curr.ptr (y + 2 * rows)[x];

        float3 vcurr_g = Rcurr * vcurr + tcurr;

        float3 vcurr_cp = Rprev_inv * (vcurr_g - tprev);         // prev camera coo space

        int2 ukr;         //projection
        ukr.x = __float2int_rn (vcurr_cp.x * intr.fx / vcurr_cp.z + intr.cx);      //4
        ukr.y = __float2int_rn (vcurr_cp.y * intr.fy / vcurr_cp.z + intr.cy);                      //4

        if (ukr.x < 0 || ukr.y < 0 || ukr.x >= cols || ukr.y >= rows || vcurr_cp.z < 0)
            return (false);

        float3 nprev_g;
        nprev_g.x = nmap_g_prev.ptr (ukr.y)[ukr.x];

        if (isnan (nprev_g.x))
            return (false);

        float3 vprev_g;
        vprev_g.x = vmap_g_prev.ptr (ukr.y       )[ukr.x];
        vprev_g.y = vmap_g_prev.ptr (ukr.y + rows)[ukr.x];
        vprev_g.z = vmap_g_prev.ptr (ukr.y + 2 * rows)[ukr.x];

        float dist = norm (vprev_g - vcurr_g);

        dist_array(x,y) = dist;
        if (dist > distThres)
            return (false);

        ncurr.y = nmap_curr.ptr (y + rows)[x];
        ncurr.z = nmap_curr.ptr (y + 2 * rows)[x];

        float3 ncurr_g = Rcurr * ncurr;

        nprev_g.y = nmap_g_prev.ptr (ukr.y + rows)[ukr.x];
        nprev_g.z = nmap_g_prev.ptr (ukr.y + 2 * rows)[ukr.x];

        float sine = norm (cross (ncurr_g, nprev_g));

        if (sine >= angleThres)
            return (false);

/*
        float resolution = RangeDistHist / nb_bin;
        int i_hist = dist / resolution;
        if (i_hist < 0 || i_hist >= nb_bin)
        {}
        else
        {
            atomicAdd(dist_PDF + i_hist, 1);
        }*/
        n = nprev_g;
        d = vprev_g;
        s = vcurr_g;
        return (true);
    }

/*

    // 1 thread only
    __device__ __forceinline__ int
    fill_CPDF_histogram() const
    {

        // int tid = Block::flattenedThreadId();
        // printf ("ThreadId for prefixSum is (should be 0, and appear once) : %d\n", tid);

        dist_CPDF[0] = dist_PDF[0];
        for (int i = 1; i < nb_bin; i++)
        {
            dist_CPDF[i] = dist_CPDF[i-1] + dist_PDF[i];
            //  printf ("   -> %d th element of CFPD : %d\n", i, dist_CPDF[i]);
        }
        return 1;
    }



    // 1000 thread
    __device__ __forceinline__ int
    estimateMedian() const
    {

        int tid = Block::flattenedThreadId();

        if (tid == 0)
        {
            int NbCorrepSoFar = dist_CPDF[tid];
            int NbCorrep_ = dist_CPDF[nb_bin-1];
            if (NbCorrepSoFar >= NbCorrep_/2)
            {
                dist_median_bin = tid;
              //  dist_median_value = tid * RangeDistHist / nb_bin;

                Output[0] = dist_CPDF [ nb_bin-1 ];
                Output[1] = dist_median_bin;
            }
        }
        else if (tid > 0)
        {
            int NbCorrepPrevious = dist_CPDF[tid-1];
            int NbCorrepSoFar = dist_CPDF[tid];
            int NbCorrep_ = dist_CPDF[nb_bin-1];
            if (NbCorrepPrevious < NbCorrep_/2 && NbCorrepSoFar >= NbCorrep_/2)
            {
                dist_median_bin = tid;
               // dist_median_value = tid * RangeDistHist / nb_bin;


                Output[0] = dist_CPDF [ nb_bin-1 ];
                Output[1] = dist_median_bin;

            }
        }


        return 1;

    }

*/
    __device__ __forceinline__ void
    operator () () const
    {
        int x = threadIdx.x + blockIdx.x * CTA_SIZE_X;
        int y = threadIdx.y + blockIdx.y * CTA_SIZE_Y;

        float3 n, d, s;
        bool found_coresp = false;

        if (x < cols && y < rows)
            found_coresp = search (x, y, n, d, s);

        float row[7];

        if (found_coresp)
        {
            *(float3*)&row[0] = cross (s, n);
            *(float3*)&row[3] = n;
            row[6] = dot (n, d - s);
        }
        else
            row[0] = row[1] = row[2] = row[3] = row[4] = row[5] = row[6] = 0.f;


        __shared__ float_type smem[CTA_SIZE];
        int tid = Block::flattenedThreadId ();


        __shared__ int cntmem[CTA_SIZE];

        __syncthreads ();
        if (found_coresp) cntmem[tid] = 1;
        else cntmem[tid] = 0;
        __syncthreads ();

        reduce<CTA_SIZE>(cntmem);
        if (tid == 0)
            cnt_buffer[blockIdx.x + gridDim.x * blockIdx.y] = cntmem[0];


        int shift = 0;
        for (int i = 0; i < 6; ++i)        //rows
        {
#pragma unroll
            for (int j = i; j < 7; ++j)          // cols + b
            {
                __syncthreads ();
                smem[tid] = row[i] * row[j];
                __syncthreads ();

                reduce<CTA_SIZE>(smem);

                if (tid == 0)
                    gbuf.ptr (shift++)[blockIdx.x + gridDim.x * blockIdx.y] = smem[0];
            }
        }
    }
};

__global__ void
combinedKernel (const Combined cs)
{
    cs ();
}
/*
__global__ void
CPDF_Filling (const Combined cs) {
    cs.fill_CPDF_histogram();
}


__global__ void
estimateMedian (const Combined cs) {
    cs.estimateMedian();
}*/

struct TranformReduction
{
    enum
    {
        CTA_SIZE = 512,
        STRIDE = CTA_SIZE,

        B = 6, COLS = 6, ROWS = 6, DIAG = 6,
        UPPER_DIAG_MAT = (COLS * ROWS - DIAG) / 2 + DIAG,
        TOTAL = UPPER_DIAG_MAT + B,

        GRID_X = TOTAL
    };

    PtrStep<float_type> gbuf;
    int length;
    mutable int* cnt_buffer;
    mutable int* output_cnt;
  //  mutable int* cnt;
    mutable float_type* output;
    mutable int out_cnt;

    __device__ __forceinline__ void
    operator () () const
    {
        {
            const float_type *beg = gbuf.ptr (blockIdx.x);
            const float_type *end = beg + length;

            int tid = threadIdx.x;
            //printf("tid = %d\n", tid);

            float_type sum = 0.f;
            for (const float_type *t = beg + tid; t < end; t += STRIDE)
                sum += *t;

            __shared__ float_type smem[CTA_SIZE];

            smem[tid] = sum;
            __syncthreads ();

            reduce<CTA_SIZE>(smem);

            if (tid == 0)
                output[blockIdx.x] = smem[0];
        }
        {
           // int tid = Block::flattenedThreadId();
           // printf("tid : %d\n", tid);
            //printf("blockIdx.x : %d\n", blockIdx.x);
            const int* beg = &(cnt_buffer[blockIdx.x]);
            const int* end = beg + length;

            int tid = threadIdx.x;
           // printf("tid = %d\n", tid);

            int sum = 0.f;
            for (const int * t = beg + tid; t < end; t += STRIDE)
                sum += *t;

            __shared__ int smem[CTA_SIZE];

            smem[tid] = sum;
            __syncthreads ();

            reduce<CTA_SIZE>(smem); //1200 / 300 / 75

          //  cnt[0] = cnt_buffer[0];

            __syncthreads ();
           if (tid == 0)
                output_cnt[blockIdx.x] = smem[0];
        }
    }
};

__global__ void
TransformEstimatorKernel2 (const TranformReduction tr)
{
    tr ();
}
}
}


//////////////////////////////////////////////////////////////////////////////////////////////////////////////////
void
pcl::device::estimateCombined (const Mat33& Rcurr, const float3& tcurr, 
                               const MapArr& vmap_curr, const MapArr& nmap_curr,
                               const Mat33& Rprev_inv, const float3& tprev, const Intr& intr,
                               const MapArr& vmap_g_prev, const MapArr& nmap_g_prev,
                               float distThres, float angleThres,
                               DeviceArray2D<float_type>& gbuf, DeviceArray<float_type>& mbuf,
                               float_type* matrixA_host, float_type* vectorB_host)
{
    int cols = vmap_curr.cols ();
    int rows = vmap_curr.rows () / 3;


  /*  //PDF and CPDF of distances
    int nbBin = 1000;
    float RangeDistHist = 0.5f;

    pcl::gpu::DeviceArray<int> dist_PDF;
    pcl::gpu::DeviceArray<int> dist_CPDF;
    int dist_PDF_host[nbBin];
    int dist_CPDF_host[nbBin];
    for (int i = 0; i < nbBin; i++)
    {
        dist_PDF_host[i] = 0;
        dist_CPDF_host[i] = 0;
    }

    dist_PDF.upload(dist_PDF_host, nbBin);
    dist_CPDF.upload(dist_CPDF_host, nbBin);


    pcl::gpu::DeviceArray<int> output_device;
    output_device.create(2);

    // Resutls
    int output[2];
    output[0] = 1;
    output[1] = 1;
    output_device.upload(output,2);
*/


    Combined cs;

    cs.Rcurr = Rcurr;
    cs.tcurr = tcurr;

    cs.vmap_curr = vmap_curr;
    cs.nmap_curr = nmap_curr;

    cs.Rprev_inv = Rprev_inv;
    cs.tprev = tprev;

    cs.intr = intr;

    cs.vmap_g_prev = vmap_g_prev;
    cs.nmap_g_prev = nmap_g_prev;

    cs.distThres = distThres;
    cs.angleThres = angleThres;

    cs.cols = cols;
    cs.rows = rows;

/*
    // PDF and Median Stuff
    cs.nb_bin = nbBin;
    cs.dist_PDF = dist_PDF;
    cs.dist_CPDF = dist_CPDF;
    cs.RangeDistHist = RangeDistHist;
    cs.Output = output_device;
*/
    //////////////////////////////

    dim3 block (Combined::CTA_SIZE_X, Combined::CTA_SIZE_Y);
    dim3 grid (1, 1, 1);
    grid.x = divUp (cols, block.x);
    grid.y = divUp (rows, block.y);

    mbuf.create (TranformReduction::TOTAL);
    if (gbuf.rows () != TranformReduction::TOTAL || gbuf.cols () < (int)(grid.x * grid.y))
        gbuf.create (TranformReduction::TOTAL, grid.x * grid.y);



    pcl::gpu::DeviceArray<int> cnt_buffer;
    cnt_buffer.create (grid.x * grid.y);

    pcl::gpu::DeviceArray2D<float> dist_array;
    dist_array.create (rows, cols);

    cs.dist_array = dist_array;
    cs.cnt_buffer = cnt_buffer;

    cs.gbuf = gbuf;

    combinedKernel<<<grid, block>>>(cs);
    cudaSafeCall ( hipGetLastError () );
    cudaSafeCall(hipDeviceSynchronize());

 /*   ///////////////
    ///NEW 2017


    // from PCF, get CPDF
    CPDF_Filling<<<1, 1>>>(cs);

    cudaSafeCall ( hipGetLastError () );
    cudaSafeCall ( hipDeviceSynchronize ());

    // Estiamte Median on CPDF -> 1000 threads
    estimateMedian<<<1, nbBin>>>(cs);

    cudaSafeCall ( hipGetLastError () );
    cudaSafeCall ( hipDeviceSynchronize ());



    output_device.download(output);


    std::cout << "Median estimation : " << output[1]*RangeDistHist/nbBin << " of bin " << output[1] << std::endl;
    std::cout << "Percentage of matches = " << (output[0]*1.f)/(cols*rows)
              << "(" << output[0] << "/" << cols*rows << ")" << std::endl;
*/


    /////////////////////////////

    TranformReduction tr;
    tr.gbuf = gbuf;
    tr.length = grid.x * grid.y;
    tr.output = mbuf;
    tr.cnt_buffer = cnt_buffer;

    pcl::gpu::DeviceArray<int> output_cnt;
    output_cnt.create (1);
    tr.output_cnt = output_cnt;


    TransformEstimatorKernel2<<<TranformReduction::TOTAL, TranformReduction::CTA_SIZE>>>(tr);
    cudaSafeCall (hipGetLastError ());
    cudaSafeCall (hipDeviceSynchronize ());

    float_type host_data[TranformReduction::TOTAL];
    mbuf.download (host_data);



    int host_cnt[1];
    output_cnt.download(host_cnt);
    int number_matches = host_cnt[0] ;

    int shift = 0;
    for (int i = 0; i < 6; ++i)  //rows
        for (int j = i; j < 7; ++j)    // cols + b
        {
            float_type value = host_data[shift++] / (number_matches*1.0f);
            if (j == 6)       // vector b
                vectorB_host[i] = value;
            else
                matrixA_host[j * 6 + i] = matrixA_host[i * 6 + j] = value;

        }

   // printf("number_matches = %f [%d/%d]\n", (number_matches*1.0f) / (cols*rows*1.0f), number_matches, cols*rows );
}
